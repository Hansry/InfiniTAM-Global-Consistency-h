#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"

struct AllocationTempData {
	int noAllocatedVoxelEntries;
	int noAllocatedExcessEntries;
	int noVisibleEntries;
};

using namespace ITMLib::Engine;

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *noVisibleEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel, bool stopMaxW>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb,
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData,float smallestVoxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);
__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float oneOverSmallestBlockSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords);
__global__ void allocateVoxelBlocksListHHash_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType);

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries);
__global__ void setToType3HH(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries, const ITMHashEntry *hashTable);

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType,
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize, int offsetToAdd);

// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = ITMVoxelBlockHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipHostFree(allocationTempData_host));
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHashEntry));
	tmpEntry.ptr = -2;
	ITMHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);

	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}

/// @brief 根据当前视角的深度图转成空间中的voxel block后，判断voxel block是否已经在voxelAllocationList或者excessAllocationList分配，若已经分配，则更改其状态为当前可见，若还未分配，则进行分配。
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view, 
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
	Vector2i depthImgSize = view->depth->noDims;
	//场景体素的大小
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	
	//invM_d为世界坐标系到当前帧坐标系的变换矩阵Twc (其中w为给定的子地图)
	//M_d为当前帧坐标系到世界坐标系下的变换矩阵Tcw
	M_d = trackingState->pose_d->GetM(); 
	M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	
	//voxelAllocationList为分配的存储sdf的内存，大小为:voxel block的数量*voxel block的大小
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	//为了哈希冲突而额外分配的内存，用来存储sdf的内存， excessALlocationList该内存的首地址
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	
	//返回的是存储hash entry的列表hashTable，得到哈希表，通过哈希映射函数建立空间点与哈希表中的entry（ITMHashEntry数据类型）之间的关系
	//其中blockPose为空间点，hashEntry为索引到VBA内存的数据类型
	//hashIdx = hashIndex(blockPose) 
	//ITMHashEntry hashEntry = hashTable[hashIdx]
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	// noTotalEntries = SDF_BUCKET_NUM（大小需要大于Voxel Block的数量） + SDF_EXCESS_LIST_SIZE(防止哈希冲突)
	int noTotalEntries = scene->index.noTotalEntries;

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeVS.x));

	//1.0m下有多少个blocks
	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = (AllocationTempData*)allocationTempData_host;
	//返回localVBA和ExcessList中倆段内存中还可以分配的Entries的数量，也可以说是分配的内存中最新的还没有被用的内存的地址
	// _______________________________
	//|___________________|||||||||||||
	//  (这里分界线即lastFreeBlockId或者ExcessListId)
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleEntries = 0;
	ITMSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	// entriesAllocType_device存储着 对应着entriAllocType_device的某个元素 的空间点是否需要被分配或者swap in
	// 1表示在ordered part进行分配, 2表示需要在excess中进行分配
	ITMSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) {
	  setToType3 << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries);
	}
	
	/// 判断当前帧空间点是否为其分配内存或者已经被分配了，状态保存在entriesAlloType_device中
	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) useSwapping = false;
	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, entriesAllocType_device, entriesVisibleType,
			blockCoords_device);
	}

	if (useSwapping)
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);
	else
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);

	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData->noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);
}

/// @brief 通过融合给定视角的深度和颜色信息来更新voxel blocks
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	///M_d为当前帧坐标系到世界坐标系下的变换矩阵Tcw,即深度图片的坐标系到世界坐标系下的变换矩阵Td,w
	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) {
	  //Note that:calib.trafo_rgb_to_depth指的是将RGB坐标系下的空间点转到Depth坐标系下，从坐标系转换角度来看，应该是Tdepth->rgb,即Depth坐标系到RGB坐标系的变换
	  //即calib.trafo_rgb_to_depth.calib_inv = Trgb,d (这里指坐标系的变换),因此Trgb,w = Trgb,d * Td,w
	  M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;
	}
	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	if (scene->sceneParams->stopIntegratingAtMaxW)
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device<TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device<TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	else
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device<TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device<TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ResetScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(scene->index.getVolumeSize().x / cudaBlockSize.x, scene->index.getVolumeSize().y / cudaBlockSize.y, scene->index.getVolumeSize().z / cudaBlockSize.z);

	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
	else
	{
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
}

// hierarchical hash

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::ITMSceneReconstructionEngine_CUDA(void)
{
	int noLevels = ITMVoxelBlockHHash::noLevels;
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedExcessEntries_device, noLevels * sizeof(int)));

	int noTotalEntries = ITMVoxelBlockHHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::~ITMSceneReconstructionEngine_CUDA(void)
{
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(noAllocatedExcessEntries_device));

	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHHashEntry));
	tmpEntry.ptr = -3;
	ITMHHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	for (int listId = 0; listId < SDF_HASH_NO_H_LEVELS; listId++)
	{
		int startPoint = listId * SDF_EXCESS_LIST_SIZE;
		fillArrayKernel<int>(excessList_ptr + startPoint, SDF_EXCESS_LIST_SIZE);
	}

	//scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
	for (int i = 0; i < SDF_HASH_NO_H_LEVELS; i++) scene->index.SetLastFreeExcessListId(i, SDF_EXCESS_LIST_SIZE - 1);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	int noTotalEntries = scene->index.noTotalEntries;
	int *lastFreeExcessListIds = scene->index.GetLastFreeExcessListIds();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	int noLevels = ITMVoxelBlockHHash::noLevels;
	int noTotalEntriesPerLevel = ITMVoxelBlockHHash::noTotalEntriesPerLevel;

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntriesPerLevel / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeAL3(256, 1);
	dim3 gridSizeAL3((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeAL.x));

	float oneOverSmallestBlockSize = 1.0f / (smallestVoxelSize * SDF_BLOCK_SIZE);

	AllocationTempData tempData;
	tempData.noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData.noAllocatedExcessEntries = 0; //NOT TO BE USED in HHash
        tempData.noVisibleEntries = 0;
        ITMSafeCall(hipMemcpy(allocationTempData_device, &tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemcpy(noAllocatedExcessEntries_device, lastFreeExcessListIds, noLevels * sizeof(int), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemset(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) setToType3HH << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries, hashTable);

	buildHHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType,
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverSmallestBlockSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksListHHash_device << <gridSizeAL3, cudaBlockSizeAL3 >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, noAllocatedExcessEntries_device, entriesAllocType_device, entriesVisibleType, blockCoords_device);
	}

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) useSwapping = false;

	for (int level = 0; level < noLevels; ++level) {
		float voxelSize = smallestVoxelSize * (1 << level);
		int levelOffset = level * noTotalEntriesPerLevel;

		if (useSwapping)
			buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, swapStates + levelOffset, noTotalEntriesPerLevel, visibleEntryIDs, (AllocationTempData*)allocationTempData_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
		else
			buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, swapStates + levelOffset, noTotalEntriesPerLevel, visibleEntryIDs, (AllocationTempData*)allocationTempData_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
	}

	if (useSwapping)
	{
		cudaBlockSizeAL = dim3(256, 1);
		gridSizeAL = dim3((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(&tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData.noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData.noAllocatedVoxelEntries;

	ITMSafeCall(hipMemcpy(lastFreeExcessListIds, noAllocatedExcessEntries_device, noLevels * sizeof(int), hipMemcpyDeviceToHost));

	scene->index.SetLastFreeExcessListIds(lastFreeExcessListIds);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	if (scene->sceneParams->stopIntegratingAtMaxW)
		integrateIntoSceneH_device<TVoxel,true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, 
			scene->index.getIndexData(), smallestVoxelSize, mu, maxW);
	else
		integrateIntoSceneH_device<TVoxel,false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, 
			scene->index.getIndexData(), smallestVoxelSize, mu, maxW);
}

// device functions

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;
	
	if (stopMaxW) if (voxelArray[locId].w_depth == maxW) return;
//	if (approximateIntegration) if (voxelArray[locId].w_depth != 0) return;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(voxelArray[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *visibleEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = visibleEntryIDs[blockIdx.x];

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;
	if (approximateIntegration) if (localVoxelBlock[locId].w_depth != 0) return;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel, bool stopMaxW>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, 
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData, float smallestVoxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = liveEntryIDs[blockIdx.x];
	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	float localVoxelSize = smallestVoxelSize * (1 << ITMVoxelBlockHHash::GetLevelForEntry(entryId));
	globalPos.x = currentHashEntry.pos.x;
	globalPos.y = currentHashEntry.pos.y;
	globalPos.z = currentHashEntry.pos.z;
	globalPos *= SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;

	pt_model.x = (float)(globalPos.x + x) * localVoxelSize;
	pt_model.y = (float)(globalPos.y + y) * localVoxelSize;
	pt_model.z = (float)(globalPos.z + z) * localVoxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation, TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

/// @brief 检查深度图x,y对应的voxel block是否已经被分配(通过检查hashEntry)，若已经被分配，是否需要将其可见性进行更新
__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float oneOverSmallestBlockSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, oneOverSmallestBlockSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;
	entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

__global__ void setToType3HH(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries, const ITMHashEntry *hashTable)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;

	int ptr = hashTable[visibleEntryIDs[entryId]].ptr;
	// blocks might have disappeared due to splitting and merging
	if (ptr < -1) entriesVisibleType[visibleEntryIDs[entryId]] = 0;
	else entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

///@brief 根据voxelAllocationList存储的状态对targetIdx进行分配
///@param voxelAllocationList 存储着对应的空间点在ordered list或者excess list是否需要分配或者分配的状态
///@param blockCoords 对应的空间点三维坐标，需要对hashEntry中的成员变量pos进行赋值 
__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; 
			hashEntry.pos.y = pt_block_all.y; 
			hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			
			//在ordered list进行hash Entry的存储
			hashTable[targetIdx] = hashEntry;
		}
		break;

	case 2: //needs allocation in the excess list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&allocData->noAllocatedExcessEntries, 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			int exlOffset = excessAllocationList[exlIdx];
			
			//由于通过hash function对voxel block的位置计算得到的hash entry已经被占用，这时候需要添加offerset以找到excess list对应的地方进行存储
			//需要强调下，offset的基准是SDF_BUCKET_NUM,即hashIdx = SDF_BUCKET_NUM + hashEntry.offset - 1
			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			//在excess list进行hash Entry的存储
			hashTable[SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			//设置为可见
			entriesVisibleType[SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void allocateVoxelBlocksListHHash_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;
	ITMHashEntry hashEntry;

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
	case 3: //needs allocation, reactivate old entry
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			if (entriesAllocType[targetIdx] == 1) hashEntry.offset = 0;
			else hashEntry.offset = hashTable[targetIdx].offset;

			hashTable[targetIdx] = hashEntry;
			entriesVisibleType[targetIdx] = 1; //make entry visible
		}
		break;

	case 2: //needs allocation in the excess list
		int level = ITMVoxelBlockHHash::GetLevelForEntry(targetIdx);

		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&noAllocatedExcessEntries[level], 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			int exlOffset = excessAllocationList[level*SDF_EXCESS_LIST_SIZE + exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[level * ITMVoxelBlockHHash::noTotalEntriesPerLevel + SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[level * ITMVoxelBlockHHash::noTotalEntriesPerLevel + SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	int hashEntry_ptr = hashTable[targetIdx].ptr;

	if (entriesVisibleType[targetIdx] > 0 && hashEntry_ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType, 
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize, int offsetToAdd)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const ITMHashEntry & hashEntry = hashTable[targetIdx];

	if (hashVisibleType == 3)
	{
		bool isVisibleEnlarged, isVisible;

		if (useSwapping)
		{
			checkBlockVisibility<true>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisibleEnlarged) hashVisibleType = 0;
		} else {
			checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisible) hashVisibleType = 0;
		}
		entriesVisibleType[targetIdx] = hashVisibleType;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	if (useSwapping)
	{
		if (hashVisibleType > 0 && swapStates[targetIdx].state != 2) swapStates[targetIdx].state = 1;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, &allocData->noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx + offsetToAdd;
	}

#if 0
	// "active list": blocks that have new information from depth image
	// currently not used...
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType == 1, noActiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) activeEntryIDs[offset] = targetIdx + offsetToAdd;
	}
#endif
}

template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;

