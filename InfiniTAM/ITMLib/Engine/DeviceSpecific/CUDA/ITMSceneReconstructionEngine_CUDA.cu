#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"

struct AllocationTempData {
	int noAllocatedVoxelEntries;
	int noAllocatedExcessEntries;
	int noVisibleEntries;
};

using namespace ITMLib::Engine;

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *noVisibleEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel, bool stopMaxW>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb,
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData,float smallestVoxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);
__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float oneOverSmallestBlockSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords);
__global__ void allocateVoxelBlocksListHHash_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType);

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries);
__global__ void setToType3HH(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries, const ITMHashEntry *hashTable);

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType,
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize, int offsetToAdd);

// host methods
/*
/// @brief 用来在整个地图上进行voxel decay
/// 有点类似于'decay_device',但是decayFull_device用于所有的voxels而不是只作用在可视voxel block上
template<class TVoxel>
__global__ void decayFull_device(const Vector4s *useBlockPositions,
                                 TVoxel *localVBA,
				 ITMHashEntry *hashTable,
				 int minAge,
				 int *voxelAllocationList,
				 int *lastFreeBlockId,
				 int *locks,
				 int currentFrame,
				 uchar *entriesVisibleType);
*/
template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = ITMVoxelBlockHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
	
	ITMSafeCall(hipMalloc((void**)&lastFreeBlockId_device, 1 * sizeof(int)));
	ITMSafeCall(hipMalloc(&locks_device, SDF_BUCKET_NUM * sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&allocatedBlockPositions_device, SDF_LOCAL_BLOCK_NUM * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipHostFree(allocationTempData_host));
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
	
	ITMSafeCall(hipFree(lastFreeBlockId_device));
	ITMSafeCall(hipFree(locks_device));
	ITMSafeCall(hipFree(allocatedBlockPositions_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();
        
	totalDecayedBlockCount = 0;
	// Clean up the visible frame queue used in voxel decay.
	while (! frameVisibleBlocks.empty()) {
		delete frameVisibleBlocks.front().blockCoords;
		frameVisibleBlocks.pop();
	}
	
	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHashEntry));
	tmpEntry.ptr = -2;
	ITMHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);

	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}

/// @brief 根据当前视角的深度图转成空间中的voxel block后，判断voxel block是否已经在voxelAllocationList或者excessAllocationList分配，若已经分配，则更改其状态为当前可见，若还未分配，则进行分配。
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view, 
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
	Vector2i depthImgSize = view->depth->noDims;
	//场景体素的大小
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	
	//invM_d为世界坐标系到当前帧坐标系的变换矩阵Twc (其中w为给定的子地图)
	//M_d为当前帧坐标系到世界坐标系下的变换矩阵Tcw
	M_d = trackingState->pose_d->GetM(); 
	M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	
	//voxelAllocationList为分配的存储sdf的内存，大小为:voxel block的数量*voxel block的大小
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	//为了哈希冲突而额外分配的内存，用来存储sdf的内存， excessALlocationList该内存的首地址
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	
	//返回的是存储hash entry的列表hashTable，得到哈希表，通过哈希映射函数建立空间点与哈希表中的entry（ITMHashEntry数据类型）之间的关系
	//其中blockPose为空间点，hashEntry为索引到VBA内存的数据类型
	//hashIdx = hashIndex(blockPose) 
	//ITMHashEntry hashEntry = hashTable[hashIdx]
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	// noTotalEntries = SDF_BUCKET_NUM（大小需要大于Voxel Block的数量） + SDF_EXCESS_LIST_SIZE(防止哈希冲突)
	int noTotalEntries = scene->index.noTotalEntries;
	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeVS.x));

	//1.0m下有多少个blocks
	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = (AllocationTempData*)allocationTempData_host;
	//返回localVBA和ExcessList中倆段内存中还可以分配的Entries的数量，也可以说是分配的内存中最新的还没有被用的内存的地址
	// _______________________________
	//|___________________|||||||||||||
	//  (这里分界线即lastFreeBlockId或者ExcessListId)
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleEntries = 0;
	ITMSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	// entriesAllocType_device存储着 对应着entriAllocType_device的某个元素 的空间点是否需要被分配或者swap in
	// 1表示在ordered part进行分配, 2表示需要在excess中进行分配
	ITMSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) {
	  setToType3 << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries);
	}
	
	/// 判断当前帧空间点是否为其分配内存或者已经被分配了，状态保存在entriesAlloType_device中
	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) useSwapping = false;
	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, entriesAllocType_device, entriesVisibleType,
			blockCoords_device);
	}

	if (useSwapping)
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);
	else
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);

	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData->noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);
	
	//为了进行decay
	int totalBlockCount = scene->index.getNumAllocatedVoxelBlocks();
	size_t visibleBlockCount = static_cast<size_t>(tempData->noVisibleEntries);
	size_t visibleBlockByteCount = visibleBlockCount * sizeof(int);
	
	//Keep track of the visible blocks, which will be used later by the voxel decay mechanism
	ORUtils::MemoryBlock<int> *visibleEntryIDsCopy = nullptr;
	if(visibleBlockByteCount > 0){
	  ///分配与当前可视的voxel block相同的CUDA内存
	  visibleEntryIDsCopy = new ORUtils::MemoryBlock<int>(visibleBlockByteCount, MEMORYDEVICE_CUDA);
	  ITMSafeCall(hipMemcpy(visibleEntryIDsCopy->GetData(MEMORYDEVICE_CUDA),
	                         visibleEntryIDs,
			         visibleBlockByteCount,
			         hipMemcpyDeviceToDevice));
	}
	
	VisibleBlockInfo visibleBlockInfo = {
	    visibleBlockCount, //count
	    frameIdx, //frameIdx
	    visibleEntryIDsCopy, //visibleEntry
	};
	
	frameVisibleBlocks.push(visibleBlockInfo);
	frameIdx ++;
	
	//This just returns the size of the pre-allocated buffer
	//返回预分配的voxel block的数量，即所有可以分配的voxel block的数量
	long allocatedBlocks = scene->index.getNumAllocatedVoxelBlocks();
	//This is the number of blocks we are using out of the chunk that was allocated initially on the GPU (for non swapping case).
	//返回已经分配了的voxel blocks的数量
	long usedBlocks = allocatedBlocks - scene->localVBA.lastFreeBlockId - 1;
	
	//返回所有分配的Excess Entries的大小，其中Excess Entries主要是为了预防哈希冲突
	long allocatedExcessEntries = SDF_EXCESS_LIST_SIZE;
	//返回已经使用的Excess Entries的大小
	long usedExcessEntries = allocatedExcessEntries - tempData->noAllocatedExcessEntries;
	
	if(usedBlocks > allocatedBlocks){
	   usedBlocks = allocatedBlocks;
	}
	
	if(usedExcessEntries > allocatedExcessEntries){
	   usedExcessEntries = allocatedExcessEntries;
	}
	
	//Display some memory status, useful for debugging mapping failures.
	float percentFree = 100.0f * (1.0f - static_cast<float>(usedBlocks)/allocatedBlocks);
	float allocatedSizeMB = scene->localVBA.allocatedSize * sizeof(ITMVoxel) / 1024.0f / 1024.0f;
	printf("[Visible: %6d | Used blocks (primary): %8ld/%ld (%.2f%% free)\n"
	       "Used excess list slots: %8ld/%ld | Total allocated size: %.2fMiB]\n",
	       tempData->noVisibleEntries,
	       usedBlocks,
	       allocatedBlocks,
	       percentFree,
	       usedExcessEntries,
	       allocatedExcessEntries,
	       allocatedSizeMB);
        if(scene->localVBA.lastFreeBlockId < 0){
	   throw std::runtime_error("Invalid free voxel block ID. InfiniTAM has run out of space in "
								 "the Voxel Block Array.");
	}
	if(scene->index.GetLastFreeExcessListId() < 0){
	   throw std::runtime_error("Invalid free excess list slot ID. InfiniTAM has run out of slots "
				    "in the hash table excess list. Consider increasing the size of "
				    "the excess list or the number of buckets.");
	}
}

/// @brief 通过融合给定视角的深度和颜色信息来更新voxel blocks
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;
	
	//如果当前视角没有任何有用的数据，则没必要进行内存分配
	if(renderState_vh->noVisibleEntries == 0){
	  return;
	}

	///M_d为当前帧坐标系到世界坐标系下的变换矩阵Tcw,即深度图片的坐标系到世界坐标系下的变换矩阵Td,w
	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) {
	  //Note that:calib.trafo_rgb_to_depth指的是将RGB坐标系下的空间点转到Depth坐标系下，从坐标系转换角度来看，应该是Tdepth->rgb,即Depth坐标系到RGB坐标系的变换
	  //即calib.trafo_rgb_to_depth.calib_inv = Trgb,d (这里指坐标系的变换),因此Trgb,w = Trgb,d * Td,w
	  M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;
	}
	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	if (scene->sceneParams->stopIntegratingAtMaxW)
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device<TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device<TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	else
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device<TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device<TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

///@brief 从hash table中将该block进行删除，同时释放block对应的VBA entry
///@param hashTable 哈希表
///@param blockPos block在voxel grid的位置，其实就是hashTable的输入，得到的是在hash entries
///@param locks Array used for locking in order to prevent data races when
///                             attempting to delete multiple elements with the same key.
///@param voxelAllocationList 分配的列表
///@param lastFreeBlockId 最新的voxel allocation list空闲处的索引
///@param entriesVisibleType For every entry in the hash table, keeps track of whether it's visible in the last/current frame.

///@Note Does not support swapping
template<class TVoxel, bool paranoid=false>
__device__ void deleteBlock(ITMHashEntry *hashTable,
                            Vector3i blockGridPos,
                            int *locks,
                            int *voxelAllocationList,
                            int *lastFreeBlockId,
                            uchar *entriesVisibleType)
{
   int keyHash = hashIndex(blockGridPos);
   //Lock the bucket for the operation, to ensure the lists stay consistent
   int status = atomicExch(&locks[keyHash], BUCKET_LOCKED);
   if(status != BUCKET_UNLOCKED){
     printf("Contention on bucket of hash value %d. Not going further with deletion of block "
	     "(%d, %d, %d).\n", keyHash, blockGridPos.x, blockGridPos.y, blockGridPos.z);
     return;
  }
  
  bool isFound = false;
  int outBlockIdx = -1;
  int outPrevBlockIdx = -1;
  findVoxel(hashTable, blockGridPos, 0, isFound, outBlockIdx, outPrevBlockIdx);
  
  bool isExcess = (outBlockIdx >= SDF_BUCKET_NUM);
  
  //Paranoid sanity checks
  if(paranoid){
    if(outPrevBlockIdx == -1){
      if(isExcess){
	printf("\n[ERROR] Found entity in excess list with no previous element (%d, %d, %d)!\n",
	       blockGridPos.x,
	       blockGridPos.y,
	       blockGridPos.z);
      }
    }
    else{
      if(!isExcess){
	printf("\n[ERROR] Found entity in bucket list with a previous guy!\n");
      }
    }
    
    if(!isFound || outBlockIdx<0){
      if(blockGridPos.x % 10 == 3){
	printf("\n\nFATAL ERROR: sanity check failed in 'decay_device' voxel (block) "
							   "found = %d, outBlockIdx = %d (%d, %d, %d) ; %s.\n",
		static_cast<int>(isFound),
	        outBlockIdx,
	        blockGridPos.x,
	        blockGridPos.y,
	        blockGridPos.z,
	        isExcess ? "excess":"non-excess"
	      );
      }
      atomicExch(&locks[keyHash], BUCKET_UNLOCKED);
      return;
    }
  }
  
  //First, deallocate the VBA slot.
  int freeListIdx = atomicAdd(&lastFreeBlockId[0],1);
  voxelAllocationList[freeListIdx+1] = hashTable[outBlockIdx].ptr;
  // TODO:Update excess freelist! (should work without doing it but leak excess slots.)
  // If updating the excess free List, make sure you also sync back the proper 'last excess slot index'.
  
  //Second, clear out the hash table entry, and do bookkeeping for buckets with more than one element.
  if(outPrevBlockIdx == -1){
    //In the ordered list
    if(hashTable[outBlockIdx].offset >= 1){
      //In the ordered list, with a successor(继承者)，继承者就是hashTable【outBlockIdx】
      long nextIdx = SDF_BUCKET_NUM + hashTable[outBlockIdx].offset -1 ;
      hashTable[outBlockIdx] = hashTable[nextIdx];
      
      entriesVisibleType[outBlockIdx] = entriesVisibleType[nextIdx];
      entriesVisibleType[nextIdx] = 0;
      
       // Free up the slot we just copied into the main VBA, in case there's still pointers
       // to it in the visible list from some to-be-decayed frame.
       // [RIP] Not doing this can mean the zombie block gets detected as valid in the future,
       // even though it's in the excess area but nobody is pointing at it.
       hashTable[nextIdx].offset = 0;
       hashTable[nextIdx].ptr = -2;
    }
    else{
       //In the ordered list, and no successor(继承者)
      hashTable[outBlockIdx].ptr = -2;
      entriesVisibleType[outBlockIdx] = 0;
    }
  }
  else{
    	// In the excess list with a successor or not.
        hashTable[outPrevBlockIdx].offset = hashTable[outBlockIdx].offset;
	hashTable[outBlockIdx].offset = 0;
	hashTable[outBlockIdx].ptr = -2;
	
	entriesVisibleType[outPrevBlockIdx] = entriesVisibleType[outBlockIdx];
	entriesVisibleType[outBlockIdx] = 0;
  }
  //Release the lock
  atomicExch(&locks[keyHash], BUCKET_UNLOCKED);
}

template<class TVoxel>
__device__ void decayVoxel(
		Vector3i blockGridPos,
		int locId,
		TVoxel *localVBA,			// could wrap in HashMap struct
		ITMHashEntry *hashTable,		// could wrap
		int minAge,
		int maxWeight,
		int *voxelAllocationList,		// could wrap
		int *lastFreeBlockId,			// could wrap
		int *locks,
		int currentFrame,
		uchar *entriesVisibleType		// could wrap
) {
	bool isFound = false;
	int blockHashIdx = -1;
	int blockPrevHashIdx = -1;
        
	int voxelIdx = findVoxel(hashTable, blockGridPos, locId, isFound, blockHashIdx, blockPrevHashIdx);

	if (-1 == blockHashIdx) {
		if (locId == 0) {
		     printf("ERROR: could not find bucket for (%d, %d, %d) @ hash ID %d.\n",
				   blockGridPos.x, blockGridPos.y, blockGridPos.z, hashIndex(blockGridPos));
		}
		return;
	}

	bool emptyVoxel = false;
	bool safeToClear = true;
	int age = currentFrame - hashTable[blockHashIdx].allocatedTime;
	if (age < minAge) {
		// Important corner case: when we had a block in the visible list, but it got deleted in
		// a previous decay pass, and ended up also getting reallocated (and thus the old ID in
		// the visible list was pointing to the wrong thing).
		safeToClear = false;
	}

	if (safeToClear) {
		// The SDF limit it EXPERIMENTAL and enabling it may be to aggressive when applied on a per-voxel basis.;
	        // localVBA[voxelIdx]为要decay的voxel
	        bool isNoisy = (localVBA[voxelIdx].w_depth <= maxWeight);
		if (isNoisy && localVBA[voxelIdx].w_depth > 0) {
			localVBA[voxelIdx].reset();
			emptyVoxel = true;
		}

		if (localVBA[voxelIdx].w_depth == 0) {
			emptyVoxel = true;
		}
	}

	// Count the empty voxels in the block, to determine if it's empty
	// TODO(andrei): Try summing all the weights and empty == weightSum < k (==3-10).
	// voxelPerrBlock为每个block中包含的voxels
	static const int voxelsPerBlock = SDF_BLOCK_SIZE3;
	__shared__ int countBuffer[voxelsPerBlock];
	countBuffer[locId] = static_cast<int>(emptyVoxel);
	__syncthreads();

	// Block-level sum for counting non-empty voxels in this block.
	// 计算该block下的非空的voxels的数量
	blockReduce(countBuffer, voxelsPerBlock, locId);
	__syncthreads();

	int emptyVoxels = countBuffer[0];
	bool emptyBlock = (emptyVoxels == voxelsPerBlock);

	if (locId == 0 && emptyBlock && safeToClear) {
		deleteBlock<TVoxel>(hashTable,
				    blockGridPos,
				    locks,
				    voxelAllocationList,
				    lastFreeBlockId,
				    entriesVisibleType);
	}
}

/// @brief 清除权重小于maxWeight的voxel blocks,同时在‘outBlocksToDellocate’中将在进程中变为空的voxel blocks标记为等待释放(pending deallocation),
///        对可视voxel block表的每一个voxel block进行decay
/// @param localVBA voxel block存储的原始内存
/// @param hashTable 将hashIdx映射到localVBA中地址的哈希表
/// @param visibleBlockPositions 通过blockIdx.x得到hashIdx，即visibleBlockPositions存储了所有voxel block的hashIdx,通过hashTable映射后可得到HashEntry
/// @param minAge 当某帧的age大于minAge,则可以考虑将该帧对应的visible voxel进行decay
/// @param maxWeight 当其对某帧的visible voxel进行decay时，若对应的voxel其depth weight小于或等于maxWeight的时候，该voxel block将会被decay
/// @param voxelAllocationList 该列表包含在localVBA使用或未被使用的voxel blocks的索引
/// @param lastFreeBlockId 'voxelAllocationList'中最后空闲未被使用(last free)的blocks的索引
/// @param locks 当删除block的时候，用来锁住bucket
/// @param currentFrame 被SLAM system处理过的当前帧的索引
/// @param entriesVisibleType 将hash table indices映射到一个enum中，该enum表示特定的voxel block是否可见
template<class TVoxel>
__global__ void decay_device(TVoxel *localVBA,
                             ITMHashEntry *hashTable,
			     int *VisibleBlockPositions,
			     int minAge,
			     int maxWeight,
			     int *voxelAllocationList,
			     int *lastFreeBlockId,
			     int *locks,
			     int currentFrame,
			     uchar *entriesVisibleType)
{
         //The local offset of the voxel in the current block
         //体素在当前voxel block的位置
         int locId = threadIdx.x + threadIdx.y * SDF_BLOCK_SIZE + threadIdx.z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
	 int entryId = VisibleBlockPositions[blockIdx.x];
	 if (entryId < 0) return;
	 
	 const ITMHashEntry &currentHashEntry = hashTable[entryId];
	 if (currentHashEntry.ptr < 0) return;
	
	///因为blockGridPos是以SDF_BLOCK_SIZE为单位的，简单的说就是其计算方法是以一个block一个block来算的，乘上SDF_BLOCK_SIZE后其单位就是1了
	Vector3i blockGridPos = currentHashEntry.pos.toInt();
	decayVoxel<TVoxel>(blockGridPos, 
	                   locId, 
	                   localVBA, 
	                   hashTable, 
	                   minAge, 
	                   maxWeight, 
	                   voxelAllocationList, 
	                   lastFreeBlockId, 
	                   locks, 
	                   currentFrame,
	                   entriesVisibleType);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::PartialDecay(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, 
										const ITMRenderState *renderState,
										const VisibleBlockInfo &visibleBlockInfo,
										int minAge,
										int maxWeight){
  
        ///创建每个scene时分配的这块内存中每个block的索引
        int *voxelAllocationList = scene->localVBA.GetAllocationList();
	///创建一个scene时用来存储voxel block时分配的内存大小，其中voxel block还存储了每个voxel的sdf值
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	
	/// cudaMemeset为local_device分配SDF_BUCKET_NUM个0
	ITMSafeCall(hipMemset(locks_device, 0, SDF_BUCKET_NUM*sizeof(int)));
	
	dim3 voxelBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(static_cast<uint32_t>(visibleBlockInfo.count));
	decay_device<TVoxel> <<< gridSize, voxelBlockSize >>> (
	         localVBA,
		 hashTable,
		 visibleBlockInfo.blockCoords->GetData(MEMORYDEVICE_CUDA),
		 minAge,
		 maxWeight,
		 voxelAllocationList,
		 lastFreeBlockId_device,
		 locks_device,
		 frameIdx,
		 ((ITMRenderState_VH*)renderState)->GetEntriesVisibleType());					       
	delete visibleBlockInfo.blockCoords;
}

///@brief 对地图进行正则化
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::Decay(
                         ITMScene<TVoxel,ITMVoxelBlockHash> *scene,
			 const ITMRenderState* renderState,
			 int maxWeight,
			 int minAge,
			 bool forceAllVoxels){
	int oldLastFreeBlockId = scene->localVBA.lastFreeBlockId;
	
	ITMSafeCall(hipMemcpy(lastFreeBlockId_device, &(scene->localVBA.lastFreeBlockId), 1*sizeof(int), hipMemcpyHostToDevice));
	
	///frameVisibleBlocks为一个queue，若其大小大于minAge,则选取最老的VisibleBlockInfo进行decay，decay后将其弹出队列
	if(static_cast<long>(frameVisibleBlocks.size()) > minAge){
	  //只是进行对‘minAge’之前帧的voxel blocks进行操作
	  VisibleBlockInfo visible = frameVisibleBlocks.front();
	  frameVisibleBlocks.pop();
	  //当要decay的当前帧的可见voxel blocks大于0时，才对该帧进行decay
	  if(visible.count > 0){
	    PartialDecay(scene, renderState, visible, minAge, maxWeight);
	  }
	}
	
	//这确保了ITM “知道” localVBA中已释放的voxel block的情况，我们需要使用它来统计释放了多少体素块
	ITMSafeCall(hipMemcpy(&(scene->localVBA.lastFreeBlockId), lastFreeBlockId_device, 1*sizeof(int), hipMemcpyDeviceToHost));
	int freeBlockCount = scene->localVBA.lastFreeBlockId - oldLastFreeBlockId;
        totalDecayedBlockCount += freeBlockCount;
	
	if(freeBlockCount > 0){
	  size_t savings = sizeof(TVoxel)*SDF_BLOCK_SIZE3*freeBlockCount;
	  float savingMb = (savings/1024.0f/1024.0f);
	  printf("Found %d candidate blocks to deallocate with weight [%d] or below and age [%d]."
	         "Saved %.2fMb. \n",
	         freeBlockCount,
	         maxWeight,
	         minAge,
	         savingMb);
	}
	else{
	  printf("Decay process found NO voxel blocks to deallocate.\n");
	} 
}

template<class TVoxel>
size_t ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::GetDecayedBlockCount() {
	return static_cast<size_t>(totalDecayedBlockCount);
}

// plain voxel array
template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ResetScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(scene->index.getVolumeSize().x / cudaBlockSize.x, scene->index.getVolumeSize().y / cudaBlockSize.y, scene->index.getVolumeSize().z / cudaBlockSize.z);

	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
	else
	{
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
}

// hierarchical hash
template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::ITMSceneReconstructionEngine_CUDA(void)
{
	int noLevels = ITMVoxelBlockHHash::noLevels;
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedExcessEntries_device, noLevels * sizeof(int)));

	int noTotalEntries = ITMVoxelBlockHHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::~ITMSceneReconstructionEngine_CUDA(void)
{
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(noAllocatedExcessEntries_device));

	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

	ITMHHashEntry tmpEntry;
	memset(&tmpEntry, 0, sizeof(ITMHHashEntry));
	tmpEntry.ptr = -3;
	ITMHHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	for (int listId = 0; listId < SDF_HASH_NO_H_LEVELS; listId++)
	{
		int startPoint = listId * SDF_EXCESS_LIST_SIZE;
		fillArrayKernel<int>(excessList_ptr + startPoint, SDF_EXCESS_LIST_SIZE);
	}

	//scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
	for (int i = 0; i < SDF_HASH_NO_H_LEVELS; i++) scene->index.SetLastFreeExcessListId(i, SDF_EXCESS_LIST_SIZE - 1);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHHashEntry *hashTable = scene->index.GetEntries();
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;

	int noTotalEntries = scene->index.noTotalEntries;
	int *lastFreeExcessListIds = scene->index.GetLastFreeExcessListIds();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	int noLevels = ITMVoxelBlockHHash::noLevels;
	int noTotalEntriesPerLevel = ITMVoxelBlockHHash::noTotalEntriesPerLevel;

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntriesPerLevel / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeAL3(256, 1);
	dim3 gridSizeAL3((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeAL.x));

	float oneOverSmallestBlockSize = 1.0f / (smallestVoxelSize * SDF_BLOCK_SIZE);

	AllocationTempData tempData;
	tempData.noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData.noAllocatedExcessEntries = 0; //NOT TO BE USED in HHash
        tempData.noVisibleEntries = 0;
        ITMSafeCall(hipMemcpy(allocationTempData_device, &tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemcpy(noAllocatedExcessEntries_device, lastFreeExcessListIds, noLevels * sizeof(int), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemset(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) setToType3HH << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries, hashTable);

	buildHHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType,
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverSmallestBlockSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksListHHash_device << <gridSizeAL3, cudaBlockSizeAL3 >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, noAllocatedExcessEntries_device, entriesAllocType_device, entriesVisibleType, blockCoords_device);
	}

	bool useSwapping = scene->useSwapping;
	if (onlyUpdateVisibleList) useSwapping = false;

	for (int level = 0; level < noLevels; ++level) {
		float voxelSize = smallestVoxelSize * (1 << level);
		int levelOffset = level * noTotalEntriesPerLevel;

		if (useSwapping)
			buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, swapStates + levelOffset, noTotalEntriesPerLevel, visibleEntryIDs, (AllocationTempData*)allocationTempData_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
		else
			buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, swapStates + levelOffset, noTotalEntriesPerLevel, visibleEntryIDs, (AllocationTempData*)allocationTempData_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
	}

	if (useSwapping)
	{
		cudaBlockSizeAL = dim3(256, 1);
		gridSizeAL = dim3((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(&tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData.noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData.noAllocatedVoxelEntries;

	ITMSafeCall(hipMemcpy(lastFreeExcessListIds, noAllocatedExcessEntries_device, noLevels * sizeof(int), hipMemcpyDeviceToHost));

	scene->index.SetLastFreeExcessListIds(lastFreeExcessListIds);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

	if (scene->sceneParams->stopIntegratingAtMaxW)
		integrateIntoSceneH_device<TVoxel,true> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, 
			scene->index.getIndexData(), smallestVoxelSize, mu, maxW);
	else
		integrateIntoSceneH_device<TVoxel,false> << <gridSize, cudaBlockSize >> >(localVBA, hashTable, visibleEntryIDs,
			rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, 
			scene->index.getIndexData(), smallestVoxelSize, mu, maxW);
}

// device functions

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;
	
	if (stopMaxW) if (voxelArray[locId].w_depth == maxW) return;
//	if (approximateIntegration) if (voxelArray[locId].w_depth != 0) return;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(voxelArray[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel, bool stopMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *visibleEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = visibleEntryIDs[blockIdx.x];

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;
	if (approximateIntegration) if (localVoxelBlock[locId].w_depth != 0) return;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel, bool stopMaxW>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, 
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData, float smallestVoxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = liveEntryIDs[blockIdx.x];
	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	float localVoxelSize = smallestVoxelSize * (1 << ITMVoxelBlockHHash::GetLevelForEntry(entryId));
	globalPos.x = currentHashEntry.pos.x;
	globalPos.y = currentHashEntry.pos.y;
	globalPos.z = currentHashEntry.pos.z;
	globalPos *= SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (stopMaxW) if (localVoxelBlock[locId].w_depth == maxW) return;

	pt_model.x = (float)(globalPos.x + x) * localVoxelSize;
	pt_model.y = (float)(globalPos.y + y) * localVoxelSize;
	pt_model.z = (float)(globalPos.z + z) * localVoxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation, TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

/// @brief 检查深度图x,y对应的voxel block是否已经被分配(通过检查hashEntry)，若已经被分配，是否需要将其可见性进行更新
__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float oneOverSmallestBlockSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, oneOverSmallestBlockSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;
	entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

__global__ void setToType3HH(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries, const ITMHashEntry *hashTable)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noVisibleEntries - 1) return;

	int ptr = hashTable[visibleEntryIDs[entryId]].ptr;
	// blocks might have disappeared due to splitting and merging
	if (ptr < -1) entriesVisibleType[visibleEntryIDs[entryId]] = 0;
	else entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

///@brief 根据voxelAllocationList存储的状态对targetIdx进行分配
///@param voxelAllocationList 存储着对应的空间点在ordered list或者excess list是否需要分配或者分配的状态
///@param blockCoords 对应的空间点三维坐标，需要对hashEntry中的成员变量pos进行赋值 
__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; 
			hashEntry.pos.y = pt_block_all.y; 
			hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;
			
			//在ordered list进行hash Entry的存储
			hashTable[targetIdx] = hashEntry;
		}
		break;

	case 2: //needs allocation in the excess list
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&allocData->noAllocatedExcessEntries, 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			ITMHashEntry hashEntry;
			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			int exlOffset = excessAllocationList[exlIdx];
			
			//由于通过hash function对voxel block的位置计算得到的hash entry已经被占用，这时候需要添加offerset以找到excess list对应的地方进行存储
			//需要强调下，offset的基准是SDF_BUCKET_NUM,即hashIdx = SDF_BUCKET_NUM + hashEntry.offset - 1
			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			//在excess list进行hash Entry的存储
			hashTable[SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			//设置为可见
			entriesVisibleType[SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void allocateVoxelBlocksListHHash_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;
	ITMHashEntry hashEntry;

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
	case 3: //needs allocation, reactivate old entry
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			if (entriesAllocType[targetIdx] == 1) hashEntry.offset = 0;
			else hashEntry.offset = hashTable[targetIdx].offset;

			hashTable[targetIdx] = hashEntry;
			entriesVisibleType[targetIdx] = 1; //make entry visible
		}
		break;

	case 2: //needs allocation in the excess list
		int level = ITMVoxelBlockHHash::GetLevelForEntry(targetIdx);

		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		exlIdx = atomicSub(&noAllocatedExcessEntries[level], 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector4s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];
			hashEntry.offset = 0;

			int exlOffset = excessAllocationList[level*SDF_EXCESS_LIST_SIZE + exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[level * ITMVoxelBlockHHash::noTotalEntriesPerLevel + SDF_BUCKET_NUM + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[level * ITMVoxelBlockHHash::noTotalEntriesPerLevel + SDF_BUCKET_NUM + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	AllocationTempData *allocData, uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	int hashEntry_ptr = hashTable[targetIdx].ptr;

	if (entriesVisibleType[targetIdx] > 0 && hashEntry_ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&allocData->noAllocatedVoxelEntries, 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
	int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType, 
	Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize, int offsetToAdd)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const ITMHashEntry & hashEntry = hashTable[targetIdx];

	if (hashVisibleType == 3)
	{
		bool isVisibleEnlarged, isVisible;

		if (useSwapping)
		{
			checkBlockVisibility<true>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisibleEnlarged) hashVisibleType = 0;
		} else {
			checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, depthImgSize);
			if (!isVisible) hashVisibleType = 0;
		}
		entriesVisibleType[targetIdx] = hashVisibleType;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	if (useSwapping)
	{
		if (hashVisibleType > 0 && swapStates[targetIdx].state != 2) swapStates[targetIdx].state = 1;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, &allocData->noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx + offsetToAdd;
	}

#if 0
	// "active list": blocks that have new information from depth image
	// currently not used...
	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType == 1, noActiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) activeEntryIDs[offset] = targetIdx + offsetToAdd;
	}
#endif
}




template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;

